
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>


// Change the code here:
// This should be changed to GPU kernel definition
void vecAdd(int numElements, const float* a, const float* b, float* c)
{
    for (int i = 0; i < numElements; i++)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    int numElements = 10000;

    float* a = (float*)calloc(numElements, sizeof(float));
    float* b = (float*)calloc(numElements, sizeof(float));
    float* c = (float*)calloc(numElements, sizeof(float));

    srand(1214134);
    for (int i = 0; i < numElements; i++)
    {
        a[i] = float(rand())/float(RAND_MAX + 1.0);
        b[i] = float(rand())/float(RAND_MAX + 1.0);
    }

    // Insert your code here:
    // 1. Create GPU device buffers
    // 2. Copy input data from host to device (vectors a and b)
    // 3. Change the CPU function call to the GPU kernel call
    vecAdd(numElements, a, b, c);
    // 4. Copy the result back (vector c)
    
    for (int i = 0; i < std::min(10, numElements); i++)
    {
        printf("%f + %f = %f\n", a[i], b[i], c[i]);
    }
    printf("...\n");

    free(a);
    free(b);
    free(c);

    // Free GPU memory here
    
    return 0;
}

